#include <hip/hip_runtime.h>

#include <stdio.h>
#include <math.h>

// CUDA kernel parameters
#define BLOCK_SIZE 256
#define MAX_NEIGHBORS 64

// Cubic spline kernel implementation on GPU
__device__ float cubic_kernel(float r, float h) {
    const float pi = 3.14159265f;
    const float h3 = h * h * h;
    const float k = 8.0f / (pi * h3);
    const float q = r / h;
    
    if (q <= 1.0f) {
        if (q <= 0.5f) {
            const float q2 = q * q;
            const float q3 = q2 * q;
            return k * (6.0f * q3 - 6.0f * q2 + 1.0f);
        } else {
            const float temp = 2.0f - q;
            return k * 2.0f * temp * temp * temp;
        }
    }
    return 0.0f;
}

// Cubic spline kernel gradient implementation on GPU
__device__ void cubic_kernel_grad(float rx, float ry, float rz, float h, float* grad_x, float* grad_y, float* grad_z) {
    const float pi = 3.14159265f;
    const float h3 = h * h * h;
    const float r = sqrtf(rx*rx + ry*ry + rz*rz);
    
    if (r > 1e-9f && r <= h) {
        const float q = r / h;
        float dW_dq = 0.0f;
        
        if (q <= 0.5f) {
            dW_dq = (8.0f / (pi * h3)) * (18.0f * q * q - 12.0f * q);
        } else if (q <= 1.0f) {
            const float temp = 2.0f - q;
            dW_dq = -(8.0f / (pi * h3)) * 6.0f * temp * temp;
        }
        
        const float factor = dW_dq / (h * r);
        *grad_x = factor * rx;
        *grad_y = factor * ry;
        *grad_z = factor * rz;
    } else {
        *grad_x = *grad_y = *grad_z = 0.0f;
    }
}

// Density computation kernel
__global__ void compute_density_kernel(
    const float* positions,
    const float* boundary_positions,
    const float* masses,
    const float* boundary_masses,
    float* densities,
    int num_particles,
    int num_boundary_particles,
    float support_radius,
    float rest_density
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_particles) return;
    
    float xi = positions[3*i];
    float yi = positions[3*i + 1];
    float zi = positions[3*i + 2];
    
    float density = 0.0f;
    
    // Density contribution from fluid particles
    for (int j = 0; j < num_particles; j++) {
        float xj = positions[3*j];
        float yj = positions[3*j + 1];
        float zj = positions[3*j + 2];
        
        float dx = xi - xj;
        float dy = yi - yj;
        float dz = zi - zj;
        float r = sqrtf(dx*dx + dy*dy + dz*dz);
        
        if (r <= support_radius) {
            float mass_j = (j < num_particles) ? masses[j] : 0.0f;
            density += mass_j * cubic_kernel(r, support_radius);
        }
    }
    
    // Density contribution from boundary particles
    for (int k = 0; k < num_boundary_particles; k++) {
        float xk = boundary_positions[3*k];
        float yk = boundary_positions[3*k + 1];
        float zk = boundary_positions[3*k + 2];
        
        float dx = xi - xk;
        float dy = yi - yk;
        float dz = zi - zk;
        float r = sqrtf(dx*dx + dy*dy + dz*dz);
        
        if (r <= support_radius) {
            float mass_k = boundary_masses[k];
            density += mass_k * cubic_kernel(r, support_radius);
        }
    }
    
    densities[i] = fmaxf(density, rest_density);
}

// Pressure computation kernel
__global__ void compute_pressure_kernel(
    const float* densities,
    float* pressures,
    int num_particles,
    float stiffness,
    float rest_density,
    float exponent
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_particles) return;
    
    float density = densities[i];
    float pressure = stiffness * (powf(density / rest_density, exponent) - 1.0f);
    pressures[i] = fmaxf(pressure, 0.0f);
}

// Force computation kernel
__global__ void compute_forces_kernel(
    const float* positions,
    const float* velocities,
    const float* densities,
    const float* pressures,
    const float* masses,
    const float* boundary_positions,
    const float* boundary_masses,
    float* accelerations,
    int num_particles,
    int num_boundary_particles,
    float support_radius,
    float viscosity
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_particles) return;
    
    float xi = positions[3*i];
    float yi = positions[3*i + 1];
    float zi = positions[3*i + 2];
    
    float vxi = velocities[3*i];
    float vyi = velocities[3*i + 1];
    float vzi = velocities[3*i + 2];
    
    float pi = pressures[i];
    float rhoi = densities[i];
    
    float ax = 0.0f, ay = 0.0f, az = 0.0f;
    
    // Force computation with fluid particles
    for (int j = 0; j < num_particles; j++) {
        if (i == j) continue;
        
        float xj = positions[3*j];
        float yj = positions[3*j + 1];
        float zj = positions[3*j + 2];
        
        float dx = xi - xj;
        float dy = yi - yj;
        float dz = zi - zj;
        float r = sqrtf(dx*dx + dy*dy + dz*dz);
        
        if (r <= support_radius && r > 1e-9f) {
            float vxj = velocities[3*j];
            float vyj = velocities[3*j + 1];
            float vzj = velocities[3*j + 2];
            
            float pj = pressures[j];
            float rhoj = densities[j];
            float massj = masses[j];
            
            // Pressure gradient
            float grad_x, grad_y, grad_z;
            cubic_kernel_grad(dx, dy, dz, support_radius, &grad_x, &grad_y, &grad_z);
            
            float pressure_factor = massj * (pi / (rhoi * rhoi) + pj / (rhoj * rhoj));
            ax -= pressure_factor * grad_x;
            ay -= pressure_factor * grad_y;
            az -= pressure_factor * grad_z;
            
            // Viscosity
            float viscosity_factor = viscosity * massj / rhoj * cubic_kernel(r, support_radius);
            ax += viscosity_factor * (vxj - vxi);
            ay += viscosity_factor * (vyj - vyi);
            az += viscosity_factor * (vzj - vzi);
        }
    }
    
    // Boundary forces
    for (int k = 0; k < num_boundary_particles; k++) {
        float xk = boundary_positions[3*k];
        float yk = boundary_positions[3*k + 1];
        float zk = boundary_positions[3*k + 2];
        
        float dx = xi - xk;
        float dy = yi - yk;
        float dz = zi - zk;
        float r = sqrtf(dx*dx + dy*dy + dz*dz);
        
        if (r <= support_radius && r > 1e-9f) {
            float massk = boundary_masses[k];
            
            float grad_x, grad_y, grad_z;
            cubic_kernel_grad(dx, dy, dz, support_radius, &grad_x, &grad_y, &grad_z);
            
            float pressure_factor = massk * pi / (rhoi * rhoi);
            ax -= pressure_factor * grad_x;
            ay -= pressure_factor * grad_y;
            az -= pressure_factor * grad_z;
        }
    }
    
    accelerations[3*i] = ax;
    accelerations[3*i + 1] = ay;
    accelerations[3*i + 2] = az;
}

// Particle integration kernel
__global__ void integrate_particles_kernel(
    float* positions,
    float* velocities,
    const float* accelerations,
    const float* gravity,
    int num_particles,
    float dt
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_particles) return;
    
    float gx = gravity[0];
    float gy = gravity[1];
    float gz = gravity[2];
    
    float ax = accelerations[3*i] + gx;
    float ay = accelerations[3*i + 1] + gy;
    float az = accelerations[3*i + 2] + gz;
    
    // Update velocity (symplectic Euler)
    velocities[3*i] += dt * ax;
    velocities[3*i + 1] += dt * ay;
    velocities[3*i + 2] += dt * az;
    
    // Update position
    positions[3*i] += dt * velocities[3*i];
    positions[3*i + 1] += dt * velocities[3*i + 1];
    positions[3*i + 2] += dt * velocities[3*i + 2];
}

// Boundary collision enforcement kernel
__global__ void enforce_boundary_collisions_kernel(
    float* positions,
    float* velocities,
    int num_particles,
    float domain_min_x, float domain_max_x,
    float domain_min_y, float domain_max_y,
    float domain_min_z, float domain_max_z,
    float damping_factor
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_particles) return;
    
    float& x = positions[3*i];
    float& y = positions[3*i + 1];
    float& z = positions[3*i + 2];
    
    float& vx = velocities[3*i];
    float& vy = velocities[3*i + 1];
    float& vz = velocities[3*i + 2];
    
    // X boundaries
    if (x < domain_min_x) {
        x = domain_min_x;
        if (vx < 0.0f) vx = -vx * damping_factor;
    } else if (x > domain_max_x) {
        x = domain_max_x;
        if (vx > 0.0f) vx = -vx * damping_factor;
    }
    
    // Y boundaries
    if (y < domain_min_y) {
        y = domain_min_y;
        if (vy < 0.0f) vy = -vy * damping_factor;
    } else if (y > domain_max_y) {
        y = domain_max_y;
        if (vy > 0.0f) vy = -vy * damping_factor;
    }
    
    // Z boundaries
    if (z < domain_min_z) {
        z = domain_min_z;
        if (vz < 0.0f) vz = -vz * damping_factor;
    } else if (z > domain_max_z) {
        z = domain_max_z;
        if (vz > 0.0f) vz = -vz * damping_factor;
    }
}

// C interface functions
extern "C" {

void cuda_compute_density(
    const float* positions,
    const float* boundary_positions,
    const float* masses,
    const float* boundary_masses,
    float* densities,
    int num_particles,
    int num_boundary_particles,
    float support_radius,
    float rest_density
) {
    dim3 blockSize(BLOCK_SIZE);
    dim3 gridSize((num_particles + blockSize.x - 1) / blockSize.x);
    
    compute_density_kernel<<<gridSize, blockSize>>>(
        positions, boundary_positions, masses, boundary_masses,
        densities, num_particles, num_boundary_particles,
        support_radius, rest_density
    );
    
    hipDeviceSynchronize();
}

void cuda_compute_pressure(
    const float* densities,
    float* pressures,
    int num_particles,
    float stiffness,
    float rest_density,
    float exponent
) {
    dim3 blockSize(BLOCK_SIZE);
    dim3 gridSize((num_particles + blockSize.x - 1) / blockSize.x);
    
    compute_pressure_kernel<<<gridSize, blockSize>>>(
        densities, pressures, num_particles,
        stiffness, rest_density, exponent
    );
    
    hipDeviceSynchronize();
}

void cuda_compute_forces(
    const float* positions,
    const float* velocities,
    const float* densities,
    const float* pressures,
    const float* masses,
    const float* boundary_positions,
    const float* boundary_masses,
    float* accelerations,
    int num_particles,
    int num_boundary_particles,
    float support_radius,
    float viscosity
) {
    dim3 blockSize(BLOCK_SIZE);
    dim3 gridSize((num_particles + blockSize.x - 1) / blockSize.x);
    
    compute_forces_kernel<<<gridSize, blockSize>>>(
        positions, velocities, densities, pressures, masses,
        boundary_positions, boundary_masses, accelerations,
        num_particles, num_boundary_particles,
        support_radius, viscosity
    );
    
    hipDeviceSynchronize();
}

void cuda_integrate_particles(
    float* positions,
    float* velocities,
    const float* accelerations,
    const float* gravity,
    int num_particles,
    float dt
) {
    dim3 blockSize(BLOCK_SIZE);
    dim3 gridSize((num_particles + blockSize.x - 1) / blockSize.x);
    
    integrate_particles_kernel<<<gridSize, blockSize>>>(
        positions, velocities, accelerations, gravity,
        num_particles, dt
    );
    
    hipDeviceSynchronize();
}

void cuda_enforce_boundary_collisions(
    float* positions,
    float* velocities,
    int num_particles,
    float domain_min_x, float domain_max_x,
    float domain_min_y, float domain_max_y,
    float domain_min_z, float domain_max_z,
    float damping_factor
) {
    dim3 blockSize(BLOCK_SIZE);
    dim3 gridSize((num_particles + blockSize.x - 1) / blockSize.x);
    
    enforce_boundary_collisions_kernel<<<gridSize, blockSize>>>(
        positions, velocities, num_particles,
        domain_min_x, domain_max_x,
        domain_min_y, domain_max_y,
        domain_min_z, domain_max_z,
        damping_factor
    );
    
    hipDeviceSynchronize();
}

} // extern "C"
